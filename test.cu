// File : test.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    if(deviceCount == 0){
        printf("no CUDA compatible GPU exitsts.\n");
    }
    else
    {
        hipDeviceProp_t pr;
        for(int i = 0; i<deviceCount;i++){
	hipGetDeviceProperties(&pr, i);
	printf("Dev #%lu is %lf \n", sizeof(char), (ceil((double)-1/2)));
	}
    }
    return 1;
}
// cache l1 size is 48kB, warp size 32
// used https://xmartlabs.github.io/cuda-calculator/ to check occupancy of SMs
// compute_sanitizer per trovare errori silenti di cuda
// max thread per multiprocessor 1024
// max registers per sm is 65536