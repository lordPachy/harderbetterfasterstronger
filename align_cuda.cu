#include "hip/hip_runtime.h"
/*
 * Exact genetic sequence alignment
 * (Using brute force)
 *
 * CUDA version
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2023/2024
 *
 * v1.3
 *
 * (c) 2024, Arturo Gonzalez-Escribano
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<limits.h>
#include<sys/time.h>
#include<stdbool.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>

/* Example of macros for error checking in CUDA */
#define CUDA_CHECK_FUNCTION( call )	{ hipError_t check = call; if ( check != hipSuccess ) fprintf(stderr, "CUDA Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }
#define CUDA_CHECK_KERNEL( )	{ hipError_t check = hipGetLastError(); if ( check != hipSuccess ) fprintf(stderr, "CUDA Kernel Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }

/* Arbitrary value to indicate that no matches are found */
#define	NOT_FOUND	-1

/* Arbitrary value to restrict the checksums period */
#define CHECKSUM_MAX	65535


/* 
 * Utils: Function to get wall time
 */
double cp_Wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}

/*
 * Utils: Random generator
 */
#include "rng.c"


/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */
/* ADD KERNELS AND OTHER FUNCTIONS HERE */
//define kernel
/*parallelize on input and patterns:
no use of shared mem
cannot handle patterns bigger than max block size
block: 1024, grid: pat_number
*/ 

	__global__ void find_patterns_v1(unsigned long *seq_len, char *sequence, char **patterns, unsigned long *pattern_found, unsigned long *pat_length){
		int pat = blockIdx.x;
		unsigned long idx = (unsigned long)((*seq_len / blockDim.x) * threadIdx.x);
		unsigned long i,j;
		//loop for paralellizing seq
		for(i = idx; i<idx+blockDim.x && i+pat_length[pat]<=*seq_len; i++){
			//check the pattern
			for(j = 0; j<pat_length[pat]; j++){
				if(sequence[i+j] != patterns[pat][j]) break;
			}
			// check if last loop ended in match
			if(j==pat_length[pat]){
				pattern_found[pat] = i;
				return;
			}
		}
	}
/*
grid = (pat_number, however many blocks I need for a pattern)
block = 1024
*/
	__global__ void find_patterns(
		unsigned long *seq_len, 
		char *sequence, 
		char **patterns, 
		unsigned long *pattern_found, 
		unsigned long *pat_length, 
		bool **g_isTheSame)
		{
		unsigned long pat = blockIdx.x;
		unsigned long idx = blockIdx.y*blockDim.x + threadIdx.x;
		unsigned long i;
		// array for reduction
		//TODO assign less memory to last block
		//unsigned long portionedShared = blockDim.x;//(pat_length[pat] - gridDim.y*blockDim.x);
		extern __shared__ bool s[];
		bool *isTheSame = s;
		bool *aggregateIsTheSame = (bool *)&isTheSame[1024];

		for(i = 0; i < *seq_len - pat_length[pat] + 1; i++){
			isTheSame[idx] = ((patterns[pat][idx] == sequence[idx+i]) && idx<pat_length[pat]);
			__syncthreads();
			

			//aggregate infrablock
			for(int r=pat_length[pat]/2; r>0; r /= 2){
				if(idx<pat_length[pat]){
					isTheSame[threadIdx.x] *=  isTheSame[threadIdx.x + r];
				}
				__syncthreads();
			} 
			
			//aggregate interblock
			if(threadIdx.x == 0) g_isTheSame[pat][blockIdx.y] = isTheSame[threadIdx.x];
			if(blockIdx.y == 0){
				if(threadIdx.x < gridDim.y){
					aggregateIsTheSame[threadIdx.x] =  g_isTheSame[pat][threadIdx.x];
				}
				__syncthreads();
					
				for(int r=gridDim.y/2; r>0; r /= 2){
					if(threadIdx.x < gridDim.y){
						aggregateIsTheSame[threadIdx.x] *= aggregateIsTheSame[threadIdx.x + r];
					}
					__syncthreads();
				}
				if(threadIdx.x == 0 && aggregateIsTheSame[0]==1){
					pattern_found[pat] = i;
					return;
				}
				// if another block has already found pattern shut down all blocks
				// looking for the same pattern
				//if(pattern_found[pat] != NOT_FOUND){
				//	return;
				//}
			}
		}
	}

/*
grid = (pat_number, however many blocks I need for a pattern, how many fractions of the input to process at the same time)
block = 1024
*/
	__global__ void find_patterns_v2_5(
		unsigned long *seq_len, 
		char *sequence, 
		char **patterns, 
		unsigned long *pattern_found, 
		unsigned long *pat_length, 
		bool **g_isTheSame)
		{
		unsigned long pat = blockIdx.x;
		unsigned long idx = blockIdx.y*blockDim.x + threadIdx.x;
		unsigned long sect = (unsigned long)((double)*pat_length/gridDim.z);
		unsigned long i;
		// array for reduction
		//TODO assign less memory to last block
		//unsigned long portionedShared = blockDim.x;//(pat_length[pat] - gridDim.y*blockDim.x);
		extern __shared__ bool s[];
		bool *isTheSame = s;
		bool *aggregateIsTheSame = (bool *)&isTheSame[1024];
		
		for(i = sect*blockIdx.z; i < (sect*(blockIdx.z+1)) && i < *seq_len - pat_length[pat] + 1; i++){
			isTheSame[idx] = ((patterns[pat][idx] == sequence[idx+i]) && idx<pat_length[pat]);
			__syncthreads();
			

			//aggregate infrablock
			for(int r=pat_length[pat]/2; r>0; r /= 2){
				if(idx<pat_length[pat]){
					isTheSame[threadIdx.x] *=  isTheSame[threadIdx.x + r];
				}
				__syncthreads();
			} 
			
			//aggregate interblock
			if(threadIdx.x == 0) g_isTheSame[pat][blockIdx.y] = isTheSame[threadIdx.x];
			if(blockIdx.y == 0){
				if(threadIdx.x < gridDim.y){
					aggregateIsTheSame[threadIdx.x] =  g_isTheSame[pat][threadIdx.x];
				}
				__syncthreads();
					
				for(int r=gridDim.y/2; r>0; r /= 2){
					if(threadIdx.x < gridDim.y){
						aggregateIsTheSame[threadIdx.x] *= aggregateIsTheSame[threadIdx.x + r];
					}
					__syncthreads();
				}
				if(threadIdx.x == 0 && aggregateIsTheSame[0]==1){
					pattern_found[pat] = i;
					return;
				}
				// if another block has already found pattern shut down all blocks
				// looking for the same pattern
				//if(pattern_found[pat] != NOT_FOUND){
				//	return;
				//}
			}
		}
	}

/*
 * Function: Increment the number of pattern matches on the sequence positions
 * 	This function can be changed and/or optimized by the students
 */
void increment_matches( int pat, unsigned long *pat_found, unsigned long *pat_length, int *seq_matches ) {
	unsigned long ind;	
	for( ind=0; ind<pat_length[pat]; ind++) {
		if ( seq_matches[ pat_found[pat] + ind ] == NOT_FOUND )
			seq_matches[ pat_found[pat] + ind ] = 0;
		else
			seq_matches[ pat_found[pat] + ind ] ++;
	}
}
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

/*
 * Function: Allocate new patttern
 */
char *pattern_allocate( rng_t *random, unsigned long pat_rng_length_mean, unsigned long pat_rng_length_dev, unsigned long seq_length, unsigned long *new_length ) {

	/* Random length */
	unsigned long length = (unsigned long)rng_next_normal( random, (double)pat_rng_length_mean, (double)pat_rng_length_dev );
	if ( length > seq_length ) length = seq_length;
	if ( length <= 0 ) length = 1;

	/* Allocate pattern */
	char *pattern = (char *)malloc( sizeof(char) * length );
	if ( pattern == NULL ) {
		fprintf(stderr,"\n-- Error allocating a pattern of size: %lu\n", length );
		exit( EXIT_FAILURE );
	}

	/* Return results */
	*new_length = length;
	return pattern;
}

/*
 * Function: Fill random sequence or pattern
 */
void generate_rng_sequence( rng_t *random, float prob_G, float prob_C, float prob_A, char *seq, unsigned long length) {
	unsigned long ind; 
	for( ind=0; ind<length; ind++ ) {
		double prob = rng_next( random );
		if( prob < prob_G ) seq[ind] = 'G';
		else if( prob < prob_C ) seq[ind] = 'C';
		else if( prob < prob_A ) seq[ind] = 'A';
		else seq[ind] = 'T';
	}
}

/*
 * Function: Copy a sample of the sequence
 */
void copy_sample_sequence( rng_t *random, char *sequence, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Copy sample */
	unsigned long ind; 
	for( ind=0; ind<length; ind++ )
		pattern[ind] = sequence[ind+location];
}

/*
 * Function: Regenerate a sample of the sequence
 */
void generate_sample_sequence( rng_t *random, rng_t random_seq, float prob_G, float prob_C, float prob_A, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length ) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Regenerate sample */
	rng_t local_random = random_seq;
	rng_skip( &local_random, location );
	generate_rng_sequence( &local_random, prob_G, prob_C, prob_A, pattern, length);
}


/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s ", program_name );
	fprintf(stderr,"<seq_length> <prob_G> <prob_C> <prob_A> <pat_rng_num> <pat_rng_length_mean> <pat_rng_length_dev> <pat_samples_num> <pat_samp_length_mean> <pat_samp_length_dev> <pat_samp_loc_mean> <pat_samp_loc_dev> <pat_samp_mix:B[efore]|A[fter]|M[ixed]> <long_seed>\n");
	fprintf(stderr,"\n");
}



/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	/* 0. Default output and error without buffering, forces to write immediately */
	setbuf(stdout, NULL);
	setbuf(stderr, NULL);

	/* 1. Read scenary arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc < 15) {
		fprintf(stderr, "\n-- Error: Not enough arguments when reading configuration from the command line\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	/* 1.2. Read argument values */
	unsigned long seq_length = atol( argv[1] );
	float prob_G = atof( argv[2] );
	float prob_C = atof( argv[3] );
	float prob_A = atof( argv[4] );
	if ( prob_G + prob_C + prob_A > 1 ) {
		fprintf(stderr, "\n-- Error: The sum of G,C,A,T nucleotid probabilities cannot be higher than 1\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}
	prob_C += prob_G;
	prob_A += prob_C;

	int pat_rng_num = atoi( argv[5] );
	unsigned long pat_rng_length_mean = atol( argv[6] );
	unsigned long pat_rng_length_dev = atol( argv[7] );
	
	int pat_samp_num = atoi( argv[8] );
	unsigned long pat_samp_length_mean = atol( argv[9] );
	unsigned long pat_samp_length_dev = atol( argv[10] );
	unsigned long pat_samp_loc_mean = atol( argv[11] );
	unsigned long pat_samp_loc_dev = atol( argv[12] );

	char pat_samp_mix = argv[13][0];
	if ( pat_samp_mix != 'B' && pat_samp_mix != 'A' && pat_samp_mix != 'M' ) {
		fprintf(stderr, "\n-- Error: Incorrect first character of pat_samp_mix: %c\n\n", pat_samp_mix);
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	unsigned long seed = atol( argv[14] );

#ifdef DEBUG
	/* DEBUG: Print arguments */
	printf("\nArguments: seq_length=%lu\n", seq_length );
	printf("Arguments: Accumulated probabilitiy G=%f, C=%f, A=%f, T=1\n", prob_G, prob_C, prob_A );
	printf("Arguments: Random patterns number=%d, length_mean=%lu, length_dev=%lu\n", pat_rng_num, pat_rng_length_mean, pat_rng_length_dev );
	printf("Arguments: Sample patterns number=%d, length_mean=%lu, length_dev=%lu, loc_mean=%lu, loc_dev=%lu\n", pat_samp_num, pat_samp_length_mean, pat_samp_length_dev, pat_samp_loc_mean, pat_samp_loc_dev );
	printf("Arguments: Type of mix: %c, Random seed: %lu\n", pat_samp_mix, seed );
	printf("\n");
#endif // DEBUG

        CUDA_CHECK_FUNCTION( hipSetDevice(0) );

	/* 2. Initialize data structures */
	/* 2.1. Skip allocate and fill sequence */
	rng_t random = rng_new( seed );
	rng_skip( &random, seq_length );

	/* 2.2. Allocate and fill patterns */
	/* 2.2.1 Allocate main structures */
	int pat_number = pat_rng_num + pat_samp_num;
	unsigned long *pat_length = (unsigned long *)malloc( sizeof(unsigned long) * pat_number );
	char **pattern = (char **)malloc( sizeof(char*) * pat_number );
	if ( pattern == NULL || pat_length == NULL ) {
		fprintf(stderr,"\n-- Error allocating the basic patterns structures for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}

	/* 2.2.2 Allocate and initialize ancillary structure for pattern types */
	int ind;
	unsigned long lind;
	#define PAT_TYPE_NONE	0
	#define PAT_TYPE_RNG	1
	#define PAT_TYPE_SAMP	2
	char *pat_type = (char *)malloc( sizeof(char) * pat_number );
	if ( pat_type == NULL ) {
		fprintf(stderr,"\n-- Error allocating ancillary structure for pattern of size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_NONE;

	/* 2.2.3 Fill up pattern types using the chosen mode */
	switch( pat_samp_mix ) {
	case 'A':
		for( ind=0; ind<pat_rng_num; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		break;
	case 'B':
		for( ind=0; ind<pat_samp_num; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		break;
	default:
		if ( pat_rng_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		}
		else if ( pat_samp_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		}
		else if ( pat_rng_num < pat_samp_num ) {
			int interval = pat_number / pat_rng_num;
			for( ind=0; ind<pat_number; ind++ ) 
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_RNG;
				else pat_type[ind] = PAT_TYPE_SAMP;
		}
		else {
			int interval = pat_number / pat_samp_num;
			for( ind=0; ind<pat_number; ind++ ) 
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_SAMP;
				else pat_type[ind] = PAT_TYPE_RNG;
		}
	}

	/* 2.2.4 Generate the patterns */
	for( ind=0; ind<pat_number; ind++ ) {
		if ( pat_type[ind] == PAT_TYPE_RNG ) {
			pattern[ind] = pattern_allocate( &random, pat_rng_length_mean, pat_rng_length_dev, seq_length, &pat_length[ind] );
			generate_rng_sequence( &random, prob_G, prob_C, prob_A, pattern[ind], pat_length[ind] );
		}
		else if ( pat_type[ind] == PAT_TYPE_SAMP ) {
			pattern[ind] = pattern_allocate( &random, pat_samp_length_mean, pat_samp_length_dev, seq_length, &pat_length[ind] );
#define REGENERATE_SAMPLE_PATTERNS
#ifdef REGENERATE_SAMPLE_PATTERNS
			rng_t random_seq_orig = rng_new( seed );
			generate_sample_sequence( &random, random_seq_orig, prob_G, prob_C, prob_A, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#else
			copy_sample_sequence( &random, sequence, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#endif
		}
		else {
			fprintf(stderr,"\n-- Error internal: Paranoic check! A pattern without type at position %d\n", ind );
			exit( EXIT_FAILURE );
		}
	}
	free( pat_type );

	/* Allocate and move the patterns to the GPU */
	unsigned long *d_pat_length;
	char **d_pattern;
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pat_length, sizeof(unsigned long) * pat_number ) );
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pattern, sizeof(char *) * pat_number ) );

	char **d_pattern_in_host = (char **)malloc( sizeof(char*) * pat_number );
	if ( d_pattern_in_host == NULL ) {
		fprintf(stderr,"\n-- Error allocating the patterns structures replicated in the host for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) {
		CUDA_CHECK_FUNCTION( hipMalloc( &(d_pattern_in_host[ind]), sizeof(char *) * pat_length[ind] ) );
        	CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern_in_host[ind], pattern[ind], pat_length[ind] * sizeof(char), hipMemcpyHostToDevice ) );
	}
	CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern, d_pattern_in_host, pat_number * sizeof(char *), hipMemcpyHostToDevice ) );

	/* Avoid the usage of arguments to take strategic decisions
	 * In a real case the user only has the patterns and sequence data to analize
	 */
	argc = 0;
	argv = NULL;
	pat_rng_num = 0;
	pat_rng_length_mean = 0;
	pat_rng_length_dev = 0;
	pat_samp_num = 0;
	pat_samp_length_mean = 0;
	pat_samp_length_dev = 0;
	pat_samp_loc_mean = 0;
	pat_samp_loc_dev = 0;
	pat_samp_mix = '0';

	/* 2.3. Other result data and structures */
	int pat_matches = 0;

	/* 2.3.1. Other results related to patterns */
	unsigned long *pat_found;
	pat_found = (unsigned long *)malloc( sizeof(unsigned long) * pat_number );
	if ( pat_found == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux pattern structure for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	
	/* 3. Start global timer */
        CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */
	/* 2.1. Allocate and fill sequence */
	char *sequence = (char *)malloc( sizeof(char) * seq_length );
	if ( sequence == NULL ) {
		fprintf(stderr,"\n-- Error allocating the sequence for size: %lu\n", seq_length );
		exit( EXIT_FAILURE );
	}

	random = rng_new( seed );
    //TODO: generate sequence using cuda
	generate_rng_sequence( &random, prob_G, prob_C, prob_A, sequence, seq_length);

#ifdef DEBUG
	/* DEBUG: Print sequence and patterns */
	printf("-----------------\n");
	printf("Sequence: ");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( "%c", sequence[lind] );
	printf("\n-----------------\n");
	printf("Patterns: %d ( rng: %d, samples: %d )\n", pat_number, pat_rng_num, pat_samp_num );
	int debug_pat;
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( "Pat[%d]: ", debug_pat );
		for( lind=0; lind<pat_length[debug_pat]; lind++ ) 
			printf( "%c", pattern[debug_pat][lind] );
		printf("\n");
	}
	printf("-----------------\n\n");
#endif // DEBUG

	/* 2.3.2. Other results related to the main sequence */
	int *seq_matches;
	seq_matches = (int *)malloc( sizeof(int) * seq_length );
	if ( seq_matches == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux sequence structures for size: %lu\n", seq_length );
		exit( EXIT_FAILURE );
	}

	/* 4. Initialize ancillary structures */
	for( ind=0; ind<pat_number; ind++) {
		pat_found[ind] = (unsigned long)NOT_FOUND;
	}
	for( lind=0; lind<seq_length; lind++) {
		seq_matches[lind] = NOT_FOUND;
	}

	/* 5. Search for each pattern */
	// identify longest pattern to assign resources
	unsigned long longest = 0;
    for(int pat = 0; pat<pat_number; pat++){
		if( pat_length[pat] > longest){
			longest = pat_length[pat];
			
		}
	}
	// 1024 is max threads per block on cluster
	int block = 1024;
	int sects = 2;
	dim3 grid(pat_number,(int)ceil((double)longest/block),sects);

    char* d_sequence; 
	unsigned long *d_pat_found_cuda;
	unsigned long *d_seq_length;
	bool **d_isTheSame;

	CUDA_CHECK_FUNCTION(hipMalloc(&d_sequence, sizeof(char) * seq_length));
	CUDA_CHECK_FUNCTION(hipMalloc(&d_pat_found_cuda, sizeof(unsigned long) * pat_number));
	CUDA_CHECK_FUNCTION(hipMalloc(&d_seq_length, sizeof(unsigned long)));
	CUDA_CHECK_FUNCTION(hipMalloc(&d_isTheSame, sizeof(bool*) * pat_number));

	// manually copying nested list
	bool **host_isTheSame = (bool**)malloc(sizeof(bool*) * pat_number);
	for(int pat = 0; pat < pat_number; pat++){
		CUDA_CHECK_FUNCTION(hipMalloc(&host_isTheSame[pat], sizeof(bool) * grid.y));
	}
	CUDA_CHECK_FUNCTION(hipMemcpy( d_isTheSame, host_isTheSame, sizeof(bool*) * pat_number, hipMemcpyHostToDevice));

	CUDA_CHECK_FUNCTION(hipMemcpy( d_pat_length,  pat_length, sizeof(unsigned long) * pat_number, hipMemcpyHostToDevice));
	CUDA_CHECK_FUNCTION(hipMemcpy( d_sequence, sequence, sizeof(char) * seq_length, hipMemcpyHostToDevice));
    CUDA_CHECK_FUNCTION(hipMemcpy( d_pat_found_cuda,  pat_found, sizeof(unsigned long) * pat_number, hipMemcpyHostToDevice));
	CUDA_CHECK_FUNCTION(hipMemcpy( d_seq_length, &seq_length, sizeof(unsigned long), hipMemcpyHostToDevice));

	find_patterns<<<grid, block, 1024*sizeof(bool)+grid.y*sizeof(bool)>>>
	(d_seq_length, d_sequence, d_pattern, d_pat_found_cuda, d_pat_length, d_isTheSame);
	CUDA_CHECK_KERNEL();

	//update the result vector
	CUDA_CHECK_FUNCTION(hipMemcpy( pat_found, d_pat_found_cuda, sizeof(unsigned long) * pat_number, hipMemcpyDeviceToHost));
	for(int pat = 0; pat < pat_number; pat++){
		if ( pat_found[pat] != (unsigned long)NOT_FOUND ) {
			/* 4.2.1. Increment the number of pattern matches on the sequence positions */
			increment_matches( pat, pat_found, pat_length, seq_matches );
		}
	}


    /* 7. Check sums */
	unsigned long checksum_matches = 0;
	unsigned long checksum_found = 0;
	for( ind=0; ind < pat_number; ind++) {
		if ( pat_found[ind] != (unsigned long)NOT_FOUND )
			checksum_found = ( checksum_found + pat_found[ind] ) % CHECKSUM_MAX;
	}
	for( lind=0; lind < seq_length; lind++) {
		if ( seq_matches[lind] != NOT_FOUND )
			pat_matches ++;
			checksum_matches = ( checksum_matches + seq_matches[lind] ) % CHECKSUM_MAX;
	}

#ifdef DEBUG
	/* DEBUG: Write results */
	printf("-----------------\n");
	printf("Found start:");
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( " %lu", pat_found[debug_pat] );
	}
	printf("\n");
	printf("-----------------\n");
	printf("Matches:");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( " %d", seq_matches[lind] );
	printf("\n");
	printf("-----------------\n");
#endif // DEBUG

	/* Free local resources */	
	free( sequence );
	free( seq_matches );

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 8. Stop global timer */
        CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );
	ttotal = cp_Wtime() - ttotal;

	/* 9. Output for leaderboard */
	printf("\n");
	/* 9.1. Total computation time */
	printf("Time: %lf\n", ttotal );

	/* 9.2. Results: Statistics */
	printf("Result: %d, %lu, %lu\n\n", 
			pat_matches,
			checksum_found,
			checksum_matches );
		
	/* 10. Free resources */	
	int i;
	for( i=0; i<pat_number; i++ ) free( pattern[i] );
	free( pattern );
	free( pat_length );
	free( pat_found );

	/* 11. End */
	return 0;
}