#include "hip/hip_runtime.h"
/*
 * Exact genetic sequence alignment
 * (Using brute force)
 *
 * CUDA version
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2023/2024
 *
 * v1.3
 *
 * (c) 2024, Arturo Gonzalez-Escribano
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<limits.h>
#include<sys/time.h>
#include<stdbool.h>

/* Headers for the CUDA assignment versions */
#include<hip/hip_runtime.h>
/* Example of macros for error checking in CUDA */
#define CUDA_CHECK_FUNCTION( call )	{ hipError_t check = call; if ( check != hipSuccess ) fprintf(stderr, "CUDA Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }
#define CUDA_CHECK_KERNEL( )	{ hipError_t check = hipGetLastError(); if ( check != hipSuccess ) fprintf(stderr, "CUDA Kernel Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }

/* Arbitrary value to indicate that no matches are found */
#define	NOT_FOUND	-1

/* Arbitrary value to restrict the checksums period */
#define CHECKSUM_MAX	65535


/* 
 * Utils: Function to get wall time
 */
double cp_Wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}

/*
 * Utils: Random generator
 */
#include "rng.c"


/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */
/* ADD KERNELS AND OTHER FUNCTIONS HERE */
//define kernel

/*
parallelize on seqence and patterns:
no extensive use of shared mem
block: 1024, grid: pat_number
*/ 
	__global__ void find_patternsv1(
		unsigned long *seq_len, 
		char *sequence, 
		char **patterns, 
		unsigned long *pattern_found, 
		unsigned long *pat_length, 
		int *resp)
		{
		int pat = blockIdx.x;
		unsigned long idx = (unsigned long)((*seq_len / blockDim.x) * threadIdx.x);
		unsigned long i,j;
		//loop for paralellizing seq
		for(i = idx; i<idx+blockDim.x && i+pat_length[pat]<=*seq_len; i++){
			//check the pattern
			for(j = 0; j<pat_length[pat]; j++){
				if(sequence[i+j] != patterns[pat][j]) break;
			}
			// check if last loop ended in match
			if(j==pat_length[pat]){
				pattern_found[pat] = i;
				return;
			}
		}
	}
/*
parallelize on literals of pattern and patterns:
no extensive use of shared mem
block: 1024, grid: pat_number
*/ 
	__global__ void find_patternsv2(
		unsigned long *seq_len, 
		char *sequence, 
		char **patterns, 
		unsigned long *pattern_found, 
		unsigned long *pat_length, 
		int *resp_thread)
		{
		int pat = blockIdx.x;
		unsigned long idx = blockIdx.y*blockDim.x + threadIdx.x;

		unsigned long i;
		extern __shared__ bool s[];
		bool *isTheSame = s;

		// init isTheSame
		isTheSame[0] = true;

		for(i = 0; i < *seq_len - pat_length[pat]+1; i++){
			if(idx < pat_length[pat]){
				for(int j = 0; j < *resp_thread; j++){
					if (idx+j < pat_length[pat]){
						if(patterns[pat][idx+j] != sequence[idx+i+j]){
							isTheSame[0] = false;
						}
					}
				}
			}
			__syncthreads();

			// result
			if(threadIdx.x == 0 && isTheSame[0]==1){
				pattern_found[pat] = i;
				return;
			}
			if(threadIdx.x == 0) isTheSame[0] = true;
		}
	}

/*
parallelize on literals of pattern and patterns:
extensive use of shared mem
block: 1024, grid: pat_number
*/ 
	__global__ void find_patternsv3(
		unsigned long *seq_len, 
		char *sequence, 
		char **patterns, 
		unsigned long *pattern_found, 
		unsigned long *pat_length, 
		int *resp_thread)
		{
		int th_radius = *resp_thread;
		int pat = blockIdx.x;
		unsigned long idx = 2*th_radius*(threadIdx.x+1);
		unsigned long i;
		unsigned long th_seq_len = *seq_len;
		unsigned long th_pat_len = pat_length[pat];

		//shared memory for sequence and one pattern
		extern __device__ __shared__ char sh[];
		char *sh_seq = sh;
		char *sh_pat = (char*)&sh_seq[th_seq_len];

		__device__ __shared__ bool sh_correct;
		sh_correct = true;

		
		// copy sequence from global into shared
		for(i=0; i < th_seq_len; i+=blockDim.x){
			if(threadIdx.x+i < th_seq_len) sh_seq[threadIdx.x+i] = sequence[threadIdx.x+i];
		} 
		// copy pattern to shared
		for(i=0; i < th_pat_len; i+=blockDim.x){
			if(threadIdx.x+i < th_pat_len) sh_pat[threadIdx.x+i] = patterns[pat][threadIdx.x+i];
		} 

		__syncthreads();

		for(i = 0; i < th_seq_len - th_pat_len+1; i++){
			if(idx < th_pat_len){
				for(int j = 0; j < 2*th_radius + 1; j++){
					if (idx+j < th_pat_len){
						if(sh_pat[idx+j] != sh_seq[i+idx+j]){
							sh_correct = false;
						}
					}
				}
			}
			__syncthreads();

			// result
			if(threadIdx.x == 0 && sh_correct==true){
				pattern_found[pat] = i;
				return;
			}
			if(threadIdx.x == 0) sh_correct = true;
		}
	}

/* 
parallelization on sequence without shared memory
coalescenced memory accesses
block: 256, grid: pat_number
*/
	__global__ void find_patterns(
		unsigned long seq_len, 
		char *sequence, 
		char **patterns, 
		unsigned long *pattern_found, 
		unsigned long *pat_length, 
		int resp_thread)
		{
		int pat = blockIdx.x;
		// init in shared memory and registers
		unsigned long th_seq_len = seq_len;
		unsigned long th_pat_len = pat_length[pat];
		unsigned long i,j;

		//loop for paralellizing seq
		for(i = 0; i+th_pat_len <= th_seq_len; i += blockDim.x){
			//check the pattern
			for(j = 0; j<th_pat_len; j++){
				if(sequence[threadIdx.x + i + j] != patterns[pat][j]) break;
			}
			// check if last loop ended in match
			if(j==th_pat_len){
				pattern_found[pat] = i;
				return;
			}
		}
	}
/*
 * Function: Increment the number of pattern matches on the sequence positions
 * 	This function can be changed and/or optimized by the students
 */
void increment_matches( int pat, unsigned long *pat_found, unsigned long *pat_length, int *seq_matches ) {
	unsigned long ind;	
	for( ind=0; ind<pat_length[pat]; ind++) {
		seq_matches[ pat_found[pat] + ind ] ++;
	}
}
/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

/*
 * Function: Allocate new patttern
 */
char *pattern_allocate( rng_t *random, unsigned long pat_rng_length_mean, unsigned long pat_rng_length_dev, unsigned long seq_length, unsigned long *new_length ) {

	/* Random length */
	unsigned long length = (unsigned long)rng_next_normal( random, (double)pat_rng_length_mean, (double)pat_rng_length_dev );
	if ( length > seq_length ) length = seq_length;
	if ( length <= 0 ) length = 1;

	/* Allocate pattern */
	char *pattern = (char *)malloc( sizeof(char) * length );
	if ( pattern == NULL ) {
		fprintf(stderr,"\n-- Error allocating a pattern of size: %lu\n", length );
		exit( EXIT_FAILURE );
	}

	/* Return results */
	*new_length = length;
	return pattern;
}

/*
 * Function: Fill random sequence or pattern
 */
void generate_rng_sequence( rng_t *random, float prob_G, float prob_C, float prob_A, char *seq, unsigned long length) {
	unsigned long ind; 
	for( ind=0; ind<length; ind++ ) {
		double prob = rng_next( random );
		if( prob < prob_G ) seq[ind] = 'G';
		else if( prob < prob_C ) seq[ind] = 'C';
		else if( prob < prob_A ) seq[ind] = 'A';
		else seq[ind] = 'T';
	}
}

/*
 * Function: Copy a sample of the sequence
 */
void copy_sample_sequence( rng_t *random, char *sequence, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Copy sample */
	unsigned long ind; 
	for( ind=0; ind<length; ind++ )
		pattern[ind] = sequence[ind+location];
}

/*
 * Function: Regenerate a sample of the sequence
 */
void generate_sample_sequence( rng_t *random, rng_t random_seq, float prob_G, float prob_C, float prob_A, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length ) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Regenerate sample */
	rng_t local_random = random_seq;
	rng_skip( &local_random, location );
	generate_rng_sequence( &local_random, prob_G, prob_C, prob_A, pattern, length);
}


/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s ", program_name );
	fprintf(stderr,"<seq_length> <prob_G> <prob_C> <prob_A> <pat_rng_num> <pat_rng_length_mean> <pat_rng_length_dev> <pat_samples_num> <pat_samp_length_mean> <pat_samp_length_dev> <pat_samp_loc_mean> <pat_samp_loc_dev> <pat_samp_mix:B[efore]|A[fter]|M[ixed]> <long_seed>\n");
	fprintf(stderr,"\n");
}



/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	/* 0. Default output and error without buffering, forces to write immediately */
	setbuf(stdout, NULL);
	setbuf(stderr, NULL);

	/* 1. Read scenary arguments */
	/* 1.1. Check minimum number of arguments */
	if (argc < 15) {
		fprintf(stderr, "\n-- Error: Not enough arguments when reading configuration from the command line\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	/* 1.2. Read argument values */
	unsigned long seq_length = atol( argv[1] );
	float prob_G = atof( argv[2] );
	float prob_C = atof( argv[3] );
	float prob_A = atof( argv[4] );
	if ( prob_G + prob_C + prob_A > 1 ) {
		fprintf(stderr, "\n-- Error: The sum of G,C,A,T nucleotid probabilities cannot be higher than 1\n\n");
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}
	prob_C += prob_G;
	prob_A += prob_C;

	int pat_rng_num = atoi( argv[5] );
	unsigned long pat_rng_length_mean = atol( argv[6] );
	unsigned long pat_rng_length_dev = atol( argv[7] );
	
	int pat_samp_num = atoi( argv[8] );
	unsigned long pat_samp_length_mean = atol( argv[9] );
	unsigned long pat_samp_length_dev = atol( argv[10] );
	unsigned long pat_samp_loc_mean = atol( argv[11] );
	unsigned long pat_samp_loc_dev = atol( argv[12] );

	char pat_samp_mix = argv[13][0];
	if ( pat_samp_mix != 'B' && pat_samp_mix != 'A' && pat_samp_mix != 'M' ) {
		fprintf(stderr, "\n-- Error: Incorrect first character of pat_samp_mix: %c\n\n", pat_samp_mix);
		show_usage( argv[0] );
		exit( EXIT_FAILURE );
	}

	unsigned long seed = atol( argv[14] );

#ifdef DEBUG
	/* DEBUG: Print arguments */
	printf("\nArguments: seq_length=%lu\n", seq_length );
	printf("Arguments: Accumulated probabilitiy G=%f, C=%f, A=%f, T=1\n", prob_G, prob_C, prob_A );
	printf("Arguments: Random patterns number=%d, length_mean=%lu, length_dev=%lu\n", pat_rng_num, pat_rng_length_mean, pat_rng_length_dev );
	printf("Arguments: Sample patterns number=%d, length_mean=%lu, length_dev=%lu, loc_mean=%lu, loc_dev=%lu\n", pat_samp_num, pat_samp_length_mean, pat_samp_length_dev, pat_samp_loc_mean, pat_samp_loc_dev );
	printf("Arguments: Type of mix: %c, Random seed: %lu\n", pat_samp_mix, seed );
	printf("\n");
#endif // DEBUG

        CUDA_CHECK_FUNCTION( hipSetDevice(0) );

	/* 2. Initialize data structures */
	/* 2.1. Skip allocate and fill sequence */
	rng_t random = rng_new( seed );
	rng_skip( &random, seq_length );

	/* 2.2. Allocate and fill patterns */
	/* 2.2.1 Allocate main structures */
	int pat_number = pat_rng_num + pat_samp_num;
	unsigned long *pat_length = (unsigned long *)malloc( sizeof(unsigned long) * pat_number );
	char **pattern = (char **)malloc( sizeof(char*) * pat_number );
	if ( pattern == NULL || pat_length == NULL ) {
		fprintf(stderr,"\n-- Error allocating the basic patterns structures for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}

	/* 2.2.2 Allocate and initialize ancillary structure for pattern types */
	int ind;
	unsigned long lind;
	#define PAT_TYPE_NONE	0
	#define PAT_TYPE_RNG	1
	#define PAT_TYPE_SAMP	2
	char *pat_type = (char *)malloc( sizeof(char) * pat_number );
	if ( pat_type == NULL ) {
		fprintf(stderr,"\n-- Error allocating ancillary structure for pattern of size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_NONE;

	/* 2.2.3 Fill up pattern types using the chosen mode */
	switch( pat_samp_mix ) {
	case 'A':
		for( ind=0; ind<pat_rng_num; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		break;
	case 'B':
		for( ind=0; ind<pat_samp_num; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		break;
	default:
		if ( pat_rng_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		}
		else if ( pat_samp_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		}
		else if ( pat_rng_num < pat_samp_num ) {
			int interval = pat_number / pat_rng_num;
			for( ind=0; ind<pat_number; ind++ ) 
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_RNG;
				else pat_type[ind] = PAT_TYPE_SAMP;
		}
		else {
			int interval = pat_number / pat_samp_num;
			for( ind=0; ind<pat_number; ind++ ) 
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_SAMP;
				else pat_type[ind] = PAT_TYPE_RNG;
		}
	}

	/* 2.2.4 Generate the patterns */
	for( ind=0; ind<pat_number; ind++ ) {
		if ( pat_type[ind] == PAT_TYPE_RNG ) {
			pattern[ind] = pattern_allocate( &random, pat_rng_length_mean, pat_rng_length_dev, seq_length, &pat_length[ind] );
			generate_rng_sequence( &random, prob_G, prob_C, prob_A, pattern[ind], pat_length[ind] );
		}
		else if ( pat_type[ind] == PAT_TYPE_SAMP ) {
			pattern[ind] = pattern_allocate( &random, pat_samp_length_mean, pat_samp_length_dev, seq_length, &pat_length[ind] );
#define REGENERATE_SAMPLE_PATTERNS
#ifdef REGENERATE_SAMPLE_PATTERNS
			rng_t random_seq_orig = rng_new( seed );
			generate_sample_sequence( &random, random_seq_orig, prob_G, prob_C, prob_A, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#else
			copy_sample_sequence( &random, sequence, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#endif
		}
		else {
			fprintf(stderr,"\n-- Error internal: Paranoic check! A pattern without type at position %d\n", ind );
			exit( EXIT_FAILURE );
		}
	}
	free( pat_type );

	/* Allocate and move the patterns to the GPU */
	unsigned long *d_pat_length;
	char **d_pattern;
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pat_length, sizeof(unsigned long) * pat_number ) );
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pattern, sizeof(char *) * pat_number ) );

	char **d_pattern_in_host = (char **)malloc( sizeof(char*) * pat_number );
	if ( d_pattern_in_host == NULL ) {
		fprintf(stderr,"\n-- Error allocating the patterns structures replicated in the host for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) {
		CUDA_CHECK_FUNCTION( hipMalloc( &(d_pattern_in_host[ind]), sizeof(char *) * pat_length[ind] ) );
        	CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern_in_host[ind], pattern[ind], pat_length[ind] * sizeof(char), hipMemcpyHostToDevice ) );
	}
	CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern, d_pattern_in_host, pat_number * sizeof(char *), hipMemcpyHostToDevice ) );

	/* Avoid the usage of arguments to take strategic decisions
	 * In a real case the user only has the patterns and sequence data to analize
	 */
	argc = 0;
	argv = NULL;
	pat_rng_num = 0;
	pat_rng_length_mean = 0;
	pat_rng_length_dev = 0;
	pat_samp_num = 0;
	pat_samp_length_mean = 0;
	pat_samp_length_dev = 0;
	pat_samp_loc_mean = 0;
	pat_samp_loc_dev = 0;
	pat_samp_mix = '0';

	/* 2.3. Other result data and structures */
	int pat_matches = 0;

	/* 2.3.1. Other results related to patterns */
	unsigned long *pat_found;
	pat_found = (unsigned long *)malloc( sizeof(unsigned long) * pat_number );
	if ( pat_found == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux pattern structure for size: %d\n", pat_number );
		exit( EXIT_FAILURE );
	}
	
	/* 3. Start global timer */
        CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */

	/* 2.1. Allocate and fill sequence */
	char *sequence = (char *)malloc( sizeof(char) * seq_length );
	if ( sequence == NULL ) {
		fprintf(stderr,"\n-- Error allocating the sequence for size: %lu\n", seq_length );
		exit( EXIT_FAILURE );
	}

	random = rng_new( seed );
    //TODO: generate sequence using cuda
	generate_rng_sequence( &random, prob_G, prob_C, prob_A, sequence, seq_length);

#ifdef DEBUG
	/* DEBUG: Print sequence and patterns */
	printf("-----------------\n");
	printf("Sequence: ");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( "%c", sequence[lind] );
	printf("\n-----------------\n");
	printf("Patterns: %d ( rng: %d, samples: %d )\n", pat_number, pat_rng_num, pat_samp_num );
	int debug_pat;
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( "Pat[%d]: ", debug_pat );
		for( lind=0; lind<pat_length[debug_pat]; lind++ ) 
			printf( "%c", pattern[debug_pat][lind] );
		printf("\n");
	}
	printf("-----------------\n\n");
#endif // DEBUG

	/* 2.3.2. Other results related to the main sequence */
	int *seq_matches;
	seq_matches = (int *)malloc( sizeof(int) * seq_length );
	if ( seq_matches == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux sequence structures for size: %lu\n", seq_length );
		exit( EXIT_FAILURE );
	}

	/* 4. Initialize ancillary structures */
	for( ind=0; ind<pat_number; ind++) {
		pat_found[ind] = (unsigned long)NOT_FOUND;
	}
	for( lind=0; lind<seq_length; lind++) {
		seq_matches[lind] = 0;
	}

	/* 5. Search for each pattern */
	/* 5.1 calculate dynamically grid dim*/
	// identify longest pattern to assign resources
	unsigned long longest = 0;
    for(int pat = 0; pat<pat_number; pat++){
		if( pat_length[pat] > longest){
			longest = pat_length[pat];
			
		}
	}
	// 1024 is max threads per block on cluster
	int block = 256;
	int sects = 1;
	int resp_thread = (int)ceil((double)(longest - block)/(2*block));
	dim3 grid(pat_number,sects);

	/* 5.2 allocate and copy memory to the device*/
    char *d_sequence;
	unsigned long *d_pat_found_cuda;
	bool **d_isTheSame;

	CUDA_CHECK_FUNCTION(hipMalloc(&d_sequence, sizeof(char) * seq_length));
	CUDA_CHECK_FUNCTION(hipMalloc(&d_pat_found_cuda, sizeof(unsigned long) * pat_number));
	CUDA_CHECK_FUNCTION(hipMalloc(&d_isTheSame, sizeof(bool*) * pat_number));


	// manually copying nested list
	bool **host_isTheSame = (bool**)malloc(sizeof(bool*) * pat_number);
	for(int pat = 0; pat < pat_number; pat++){
		CUDA_CHECK_FUNCTION(hipMalloc(&host_isTheSame[pat], sizeof(bool) * grid.y));
	}
	CUDA_CHECK_FUNCTION(hipMemcpy( d_isTheSame, host_isTheSame, sizeof(bool*) * pat_number, hipMemcpyHostToDevice));

	CUDA_CHECK_FUNCTION(hipMemcpy( d_pat_length,  pat_length, sizeof(unsigned long) * pat_number, hipMemcpyHostToDevice));
	CUDA_CHECK_FUNCTION(hipMemcpy( d_sequence, sequence, sizeof(char) * seq_length, hipMemcpyHostToDevice));
    CUDA_CHECK_FUNCTION(hipMemcpy( d_pat_found_cuda,  pat_found, sizeof(unsigned long) * pat_number, hipMemcpyHostToDevice));

	/* 5.3 launch the kernel */
	find_patterns<<<grid, block>>>
	(seq_length, d_sequence, d_pattern, d_pat_found_cuda, d_pat_length, resp_thread);
	hipDeviceSynchronize();
	CUDA_CHECK_KERNEL();

	/* 5.4 increase matches */
	CUDA_CHECK_FUNCTION(hipMemcpy( pat_found, d_pat_found_cuda, sizeof(unsigned long) * pat_number, hipMemcpyDeviceToHost));
	for(int pat = 0; pat < pat_number; pat++){
		if ( pat_found[pat] != (unsigned long)NOT_FOUND ) {
			/* 4.2.1. Increment the number of pattern matches on the sequence positions */
			increment_matches( pat, pat_found, pat_length, seq_matches );
			pat_matches++;
		}
	}


    /* 7. Check sums */
	unsigned long checksum_matches = 0;
	unsigned long checksum_found = 0;
	for( ind=0; ind < pat_number; ind++) {
		if ( pat_found[ind] != (unsigned long)NOT_FOUND )
			checksum_found = ( checksum_found + pat_found[ind] ) % CHECKSUM_MAX;
	}
	for( lind=0; lind < seq_length; lind++) {
		if ( seq_matches[lind] != 0 )
			checksum_matches = ( checksum_matches + seq_matches[lind] ) % CHECKSUM_MAX;
	}

#ifdef DEBUG
	/* DEBUG: Write results */
	printf("-----------------\n");
	printf("Found start:");
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( " %lu", pat_found[debug_pat] );
	}
	printf("\n");
	printf("-----------------\n");
	printf("Matches:");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( " %d", seq_matches[lind] );
	printf("\n");
	printf("-----------------\n");
#endif // DEBUG

	/* Free local resources */	
	free( sequence );
	free( seq_matches );

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 8. Stop global timer */
        CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );
	ttotal = cp_Wtime() - ttotal;

	/* 9. Output for leaderboard */
	printf("\n");
	/* 9.1. Total computation time */
	printf("Time: %lf\n", ttotal );

	/* 9.2. Results: Statistics */
	printf("Result: %d, %lu, %lu\n\n", 
			pat_matches,
			checksum_found,
			checksum_matches );

		
	/* 10. Free resources */	
	int i;
	for( i=0; i<pat_number; i++ ) free( pattern[i] );
	free( pattern );
	free( pat_length );
	free( pat_found );

	/* 11. End */
	return 0;
}